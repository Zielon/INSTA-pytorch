#include "hip/hip_runtime.h"
/*
 * Max-Planck-Gesellschaft zur Förderung der Wissenschaften e.V. (MPG) is
 * holder of all proprietary rights on this computer program.
 * You can only use this computer program if you have closed
 * a license agreement with MPG or you get the right to use the computer
 * program from someone who is authorized to grant you that right.
 * Any use of the computer program without a valid license is prohibited and
 * liable to prosecution.
 *
 * Copyright©2019 Max-Planck-Gesellschaft zur Förderung
 * der Wissenschaften e.V. (MPG). acting on behalf of its Max Planck Institute
 * for Intelligent Systems. All rights reserved.
 *
 * @author Vasileios Choutas
 * Contact: vassilis.choutas@tuebingen.mpg.de
 * Contact: ps-license@tuebingen.mpg.de
 *
 */

#include <torch/extension.h>
#include <torch/types.h>

#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/sort.h>

#include <iostream>
#include <string>
#include <type_traits>
#include <utility>
#include <vector>

#include "aabb.hpp"
#include "defs.hpp"
#include "double_vec_ops.h"
#include "hip/hip_vector_types.h"
#include "math_utils.hpp"
#include "priority_queue.hpp"
#include "triangle.hpp"

// Number of threads per block for CUDA kernel launch
#ifndef NUM_THREADS
#define NUM_THREADS 256
#endif

#ifndef FORCE_INLINE
#define FORCE_INLINE 1
#endif /* ifndef FORCE_INLINE */

#ifndef BVH_PROFILING
#define BVH_PROFILING 0
#endif /* ifndef BVH_PROFILING */

#ifndef ERROR_CHECKING
#define ERROR_CHECKING 1
#endif /* ifndef ERROR_CHECKING */

// Macro for checking cuda errors following a cuda launch or api call
#if ERROR_CHECKING == 1
#define cudaCheckError()                                                       \
  {                                                                            \
    hipDeviceSynchronize();                                                   \
    hipError_t e = hipGetLastError();                                        \
    if (e != hipSuccess) {                                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,                 \
             hipGetErrorString(e));                                           \
      exit(0);                                                                 \
    }                                                                          \
  }
#else
#define cudaCheckError()
#endif

typedef unsigned int MortonCode;

template <typename T>
std::ostream &operator<<(std::ostream &os, const vec3<T> &x) {
  os << x.x << ", " << x.y << ", " << x.z;
  return os;
}

std::ostream &operator<<(std::ostream &os, const vec3<float> &x) {
  os << x.x << ", " << x.y << ", " << x.z;
  return os;
}

std::ostream &operator<<(std::ostream &os, const vec3<double> &x) {
  os << x.x << ", " << x.y << ", " << x.z;
  return os;
}

template <typename T> std::ostream &operator<<(std::ostream &os, vec3<T> x) {
  os << x.x << ", " << x.y << ", " << x.z;
  return os;
}

__host__ __device__ inline double3 fmin(const double3 &a, const double3 &b) {
  return make_double3(fmin(a.x, b.x), fmin(a.y, b.y), fmin(a.z, b.z));
}

__host__ __device__ inline double3 fmax(const double3 &a, const double3 &b) {
  return make_double3(fmax(a.x, b.x), fmax(a.y, b.y), fmax(a.z, b.z));
}

struct is_valid_cnt : public thrust::unary_function<long2, int> {
public:
  __host__ __device__ int operator()(long2 vec) const {
    return vec.x >= 0 && vec.y >= 0;
  }
};

template <typename T>
__host__ __device__ T pointToTriangleDistance(vec3<T> p,
                                              TrianglePtr<T> tri_ptr) {
  vec3<T> a = tri_ptr->v0;
  vec3<T> b = tri_ptr->v1;
  vec3<T> c = tri_ptr->v2;

  vec3<T> ba = b - a;
  vec3<T> pa = p - a;
  vec3<T> cb = c - b;
  vec3<T> pb = p - b;
  vec3<T> ac = a - c;
  vec3<T> pc = p - c;
  vec3<T> nor = cross(ba, ac);

  return (sign<T>(dot(cross(ba, nor), pa)) + sign<T>(dot(cross(cb, nor), pb)) +
              sign<T>(dot(cross(ac, nor), pc)) <
          2.0)
             ? min(min(dot2<T>(ba * clamp(dot(ba, pa) / dot2<T>(ba), 0.0, 1.0) -
                               pa),
                       dot2<T>(cb * clamp(dot(cb, pb) / dot2<T>(cb), 0.0, 1.0) -
                               pb)),
                   dot2<T>(ac * clamp(dot(ac, pc) / dot2<T>(ac), 0.0, 1.0) -
                           pc))
             : dot(nor, pa) * dot(nor, pa) / dot2<T>(nor);
}

template <typename T>
__host__ __device__ T pointToTriangleDistance(vec3<T> p, TrianglePtr<T> tri_ptr,
                                              vec3<T> *closest_bc,
                                              vec3<T> *closest_point) {
  vec3<T> a = tri_ptr->v0;
  vec3<T> b = tri_ptr->v1;
  vec3<T> c = tri_ptr->v2;

  // Check if P in vertex region outside A
  vec3<T> ab = b - a;
  vec3<T> ac = c - a;
  vec3<T> ap = p - a;
  T d1 = dot(ab, ap);
  T d2 = dot(ac, ap);
  if (d1 <= static_cast<T>(0) && d2 <= static_cast<T>(0)) {
    *closest_point = a;
    *closest_bc = make_vec3<T>(static_cast<T>(1.0), static_cast<T>(0.0), static_cast<T>(0.0));
    return dot(ap, ap);
  }
  // Check if P in vertex region outside B
  vec3<T> bp = p - b;
  T d3 = dot(ab, bp);
  T d4 = dot(ac, bp);

  if (d3 >= 0.0f && d4 <= d3) {
    *closest_point = b;
    *closest_bc = make_vec3<T>(static_cast<T>(0.0), static_cast<T>(1.0), static_cast<T>(0.0));
    return dot(bp, bp);
  }
  // Check if P in edge region of AB, if so return projection of P onto AB
  T vc = d1 * d4 - d3 * d2;
  if (vc <= static_cast<T>(0) && d1 >= static_cast<T>(0) &&
      d3 <= static_cast<T>(0)) {
    T v = d1 / (d1 - d3);
    *closest_point = a + v * ab;
    *closest_bc = make_vec3<T>(static_cast<T>(1 - v), static_cast<T>(v), static_cast<T>(0.0));
    return dot(p - *closest_point, p - *closest_point);
  }
  // Check if P in vertex region outside C
  vec3<T> cp = p - c;
  T d5 = dot(ab, cp);
  T d6 = dot(ac, cp);
  if (d6 >= static_cast<T>(0) && d5 <= d6) {
    *closest_point = c;
    *closest_bc = make_vec3<T>(0.0, 0.0, 1.0);
    return dot(cp, cp);
  }
  // Check if P in edge region of AC, if so return projection of P onto AC
  T vb = d5 * d2 - d1 * d6;
  if (vb <= static_cast<T>(0) && d2 >= static_cast<T>(0) &&
      d6 <= static_cast<T>(0)) {
    T w = d2 / (d2 - d6);
    *closest_point = a + w * ac;
    *closest_bc = make_vec3<T>(static_cast<T>(1 - w), static_cast<T>(0.0), static_cast<T>(w));
    return dot(p - *closest_point, p - *closest_point);
  }
  // Check if P in edge region of BC, if so return projection of P onto BC
  T va = d3 * d6 - d5 * d4;
  if (va <= static_cast<T>(0) && (d4 - d3) >= static_cast<T>(0) &&
      (d5 - d6) >= static_cast<T>(0)) {
    T w = (d4 - d3) / ((d4 - d3) + (d5 - d6));
    *closest_point = b + w * (c - b);
    *closest_bc = make_vec3<T>(static_cast<T>(0), static_cast<T>(1 - w), static_cast<T>(w));
    return dot(p - *closest_point, p - *closest_point);
  }
  // P inside face region. Compute Q through its barycentric coordinates (u,v,w)
  T denom = static_cast<T>(1) / (va + vb + vc);
  T v = vb * denom;
  T w = vc * denom;
  *closest_point = a + v * ab + w * ac;
  *closest_bc = make_vec3<T>(static_cast<T>(1 - v - w), static_cast<T>(v), static_cast<T>(w));
  return dot(p - *closest_point, p - *closest_point);
}

template <typename T>
__global__ void ComputeTriBoundingBoxes(Triangle<T> *triangles,
                                        int num_triangles, AABB<T> *bboxes) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < num_triangles;
       idx += blockDim.x * gridDim.x) {
    bboxes[idx] = triangles[idx].ComputeBBox();
  }
  return;
}

template <typename T> struct BVHNode {
public:
  AABB<T> bbox;

  // __host__ __device__
  // BVHNode(): left(nullptr), right(nullptr), tri_ptr(nullptr), idx(-1);

  TrianglePtr<T> tri_ptr;
  BVHNode<T> *left;
  BVHNode<T> *right;
  BVHNode<T> *parent;
  __host__ __device__ inline bool isLeaf() { return !left && !right; };

  // The index of the object contained in the node
  int idx;
};

template <typename T> using BVHNodePtr = BVHNode<T> *;

template <typename T>
__device__
#if FORCE_INLINE == 1
    __forceinline__
#endif
    bool
    checkOverlap(const AABB<T> &bbox1, const AABB<T> &bbox2) {
  return (bbox1.min_t.x <= bbox2.max_t.x) && (bbox1.max_t.x >= bbox2.min_t.x) &&
         (bbox1.min_t.y <= bbox2.max_t.y) && (bbox1.max_t.y >= bbox2.min_t.y) &&
         (bbox1.min_t.z <= bbox2.max_t.z) && (bbox1.max_t.z >= bbox2.min_t.z);
}

template <typename T, int StackSize = 32>
__device__ T traverseBVHStack(const vec3<T> &queryPoint, BVHNodePtr<T> root,
                              long *closest_face, vec3<T> *closest_bc,
                              vec3<T> *closestPoint) {
  BVHNodePtr<T> stack[StackSize];
  BVHNodePtr<T> *stackPtr = stack;
  *stackPtr++ = nullptr; // push

  BVHNodePtr<T> node = root;
  T closest_distance = std::is_same<T, float>::value ? FLT_MAX : DBL_MAX;

  do {
    // Check each child node for overlap.
    BVHNodePtr<T> childL = node->left;
    BVHNodePtr<T> childR = node->right;

    T distance_left = pointToAABBDistance<T>(queryPoint, childL->bbox);
    T distance_right = pointToAABBDistance<T>(queryPoint, childR->bbox);

    bool checkL = distance_left <= closest_distance;
    bool checkR = distance_right <= closest_distance;

    if (checkL && childL->isLeaf()) {
      // If  the child is a leaf then
      TrianglePtr<T> tri_ptr = childL->tri_ptr;
      vec3<T> curr_clos_point;
      vec3<T> curr_closest_bc;

      T distance_left = pointToTriangleDistance<T>(
          queryPoint, tri_ptr, &curr_closest_bc, &curr_clos_point);
      if (distance_left <= closest_distance) {
        closest_distance = distance_left;
        *closest_face = childL->idx;
        *closestPoint = curr_clos_point;
        *closest_bc = curr_closest_bc;
      }
    }

    if (checkR && childR->isLeaf()) {
      // If  the child is a leaf then
      TrianglePtr<T> tri_ptr = childR->tri_ptr;
      vec3<T> curr_clos_point;
      vec3<T> curr_closest_bc;

      T distance_right = pointToTriangleDistance<T>(
          queryPoint, tri_ptr, &curr_closest_bc, &curr_clos_point);
      if (distance_right <= closest_distance) {
        closest_distance = distance_right;
        *closest_face = childR->idx;
        *closestPoint = curr_clos_point;
        *closest_bc = curr_closest_bc;
      }
    }
    // Query overlaps an internal node => traverse.
    bool traverseL = (checkL && !childL->isLeaf());
    bool traverseR = (checkR && !childR->isLeaf());

    if (!traverseL && !traverseR) {
      node = *--stackPtr; // pop
    } else {
      node = (traverseL) ? childL : childR;
      if (traverseL && traverseR) {
        *stackPtr++ = childR; // push
      }
    }
  } while (node != nullptr);

  return closest_distance;
}

template <typename T, int QueueSize = 32>
__device__ T traverseBVH(const vec3<T> &queryPoint, BVHNodePtr<T> root,
                         long *closest_face, vec3<T> *closest_bc,
                         vec3<T> *closestPoint) {
  // Create a priority queue
  PriorityQueue<T, BVHNodePtr<T>, QueueSize> queue;

  T root_dist = pointToAABBDistance(queryPoint, root->bbox);

  queue.insert_key(root_dist, root);

  BVHNodePtr<T> node = nullptr;

  T closest_distance = std::is_same<T, float>::value ? FLT_MAX : DBL_MAX;

  while (queue.get_size() > 0) {
    std::pair<T, BVHNodePtr<T>> output = queue.extract();
    // T curr_distance = output.first;
    node = output.second;

    // Check each child node for overlap.
    BVHNodePtr<T> childL = node->left;
    BVHNodePtr<T> childR = node->right;

    T distance_left = pointToAABBDistance<T>(queryPoint, childL->bbox);
    T distance_right = pointToAABBDistance<T>(queryPoint, childR->bbox);

    if (distance_left <= closest_distance) {
      if (childL->isLeaf()) {
        // If  the child is a leaf then
        TrianglePtr<T> tri_ptr = childL->tri_ptr;
        vec3<T> curr_clos_point;
        vec3<T> curr_closest_bc;

        T distance_left = pointToTriangleDistance<T>(
            queryPoint, tri_ptr, &curr_closest_bc, &curr_clos_point);
        if (distance_left <= closest_distance) {
          closest_distance = distance_left;
          *closest_face = childL->idx;
          *closestPoint = curr_clos_point;
          *closest_bc = curr_closest_bc;
        }
      } else {
        queue.insert_key(distance_left, childL);
      }
    }

    if (distance_right <= closest_distance) {
      if (childR->isLeaf()) {
        // If  the child is a leaf then
        TrianglePtr<T> tri_ptr = childR->tri_ptr;
        vec3<T> curr_clos_point;
        vec3<T> curr_closest_bc;

        T distance_right = pointToTriangleDistance<T>(
            queryPoint, tri_ptr, &curr_closest_bc, &curr_clos_point);
        if (distance_right <= closest_distance) {
          closest_distance = distance_right;
          *closest_face = childR->idx;
          *closestPoint = curr_clos_point;
          *closest_bc = curr_closest_bc;
        }
      } else {
        queue.insert_key(distance_right, childR);
      }
    }
  }

  return closest_distance;
}

template <typename T, int QueueSize = 32>
__global__ void findNearestNeighbor(vec3<T> *query_points, T *distances,
                                    vec3<T> *closest_points,
                                    long *closest_faces,
                                    vec3<T> *closest_bcs,
                                    BVHNodePtr<T> root, int num_points,
                                    bool use_stack = true) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_points;
       idx += blockDim.x * gridDim.x) {
    vec3<T> query_point = query_points[idx];

    long closest_face;
    vec3<T> closest_bc;
    vec3<T> closest_point;

    T closest_distance;
    if (use_stack) {
      closest_distance = traverseBVHStack<T, QueueSize>(
          query_point, root, &closest_face, &closest_bc, &closest_point);
    } else {
      closest_distance = traverseBVH<T, QueueSize>(
          query_point, root, &closest_face, &closest_bc, &closest_point);
    }
    distances[idx] = closest_distance;
    closest_points[idx] = closest_point;
    closest_faces[idx] = closest_face;
    closest_bcs[idx] = closest_bc;
  }
  return;
}

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__
#if FORCE_INLINE == 1
    __forceinline__
#endif
        MortonCode
        expandBits(MortonCode v) {
  // Shift 16
  v = (v * 0x00010001u) & 0xFF0000FFu;
  // Shift 8
  v = (v * 0x00000101u) & 0x0F00F00Fu;
  // Shift 4
  v = (v * 0x00000011u) & 0xC30C30C3u;
  // Shift 2
  v = (v * 0x00000005u) & 0x49249249u;
  return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
template <typename T>
__device__
#if FORCE_INLINE == 1
    __forceinline__
#endif
        MortonCode
        morton3D(T x, T y, T z) {
  x = min(max(x * 1024.0f, 0.0f), 1023.0f);
  y = min(max(y * 1024.0f, 0.0f), 1023.0f);
  z = min(max(z * 1024.0f, 0.0f), 1023.0f);
  MortonCode xx = expandBits((MortonCode)x);
  MortonCode yy = expandBits((MortonCode)y);
  MortonCode zz = expandBits((MortonCode)z);
  return xx * 4 + yy * 2 + zz;
}

template <typename T>
__global__ void ComputePointMortonCodes(vec3<T> *points, vec3<T> *in_points,
                                        int num_points,
                                        MortonCode *morton_codes) {
  AABB<T> scene_bb(-1, -1, -1, 1, 1, 1);
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < num_points;
       idx += blockDim.x * gridDim.x) {
    // Fetch the current triangle
    vec3<T> point = in_points[idx];

    T x = (point.x - scene_bb.min_t.x) / (scene_bb.max_t.x - scene_bb.min_t.x);
    T y = (point.y - scene_bb.min_t.y) / (scene_bb.max_t.y - scene_bb.min_t.y);
    T z = (point.z - scene_bb.min_t.z) / (scene_bb.max_t.z - scene_bb.min_t.z);

    morton_codes[idx] = morton3D<T>(x, y, z);
    points[idx] = point;
  }
  return;
}

template <typename T>
__global__ void ComputeMortonCodes(Triangle<T> *triangles, int num_triangles,
                                   AABB<T> *scene_bb,
                                   MortonCode *morton_codes) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < num_triangles;
       idx += blockDim.x * gridDim.x) {
    // Fetch the current triangle
    Triangle<T> tri = triangles[idx];
    vec3<T> centroid = (tri.v0 + tri.v1 + tri.v2) / (T)3.0;

    T x = (centroid.x - scene_bb->min_t.x) /
          (scene_bb->max_t.x - scene_bb->min_t.x);
    T y = (centroid.y - scene_bb->min_t.y) /
          (scene_bb->max_t.y - scene_bb->min_t.y);
    T z = (centroid.z - scene_bb->min_t.z) /
          (scene_bb->max_t.z - scene_bb->min_t.z);

    morton_codes[idx] = morton3D<T>(x, y, z);
  }
  return;
}

__device__
#if FORCE_INLINE == 1
    __forceinline__
#endif
    int
    LongestCommonPrefix(int i, int j, MortonCode *morton_codes,
                        int num_triangles, int *triangle_ids) {
  // This function will be called for i - 1, i, i + 1, so we might go beyond
  // the array limits
  if (i < 0 || i > num_triangles - 1 || j < 0 || j > num_triangles - 1)
    return -1;

  MortonCode key1 = morton_codes[i];
  MortonCode key2 = morton_codes[j];

  if (key1 == key2) {
    // Duplicate key:__clzll(key1 ^ key2) will be equal to the number of
    // bits in key[1, 2]. Add the number of leading zeros between the
    // indices
    return __clz(key1 ^ key2) + __clz(triangle_ids[i] ^ triangle_ids[j]);
  } else {
    // Keys are different
    return __clz(key1 ^ key2);
  }
}

template <typename T>
__global__ void BuildRadixTree(MortonCode *morton_codes, int num_triangles,
                               int *triangle_ids, BVHNodePtr<T> internal_nodes,
                               BVHNodePtr<T> leaf_nodes) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < num_triangles - 1;
       idx += blockDim.x * gridDim.x) {
    // if (idx >= num_triangles - 1)
    // return;

    int delta_next = LongestCommonPrefix(idx, idx + 1, morton_codes,
                                         num_triangles, triangle_ids);
    int delta_last = LongestCommonPrefix(idx, idx - 1, morton_codes,
                                         num_triangles, triangle_ids);
    // Find the direction of the range
    int direction = delta_next - delta_last >= 0 ? 1 : -1;

    int delta_min = LongestCommonPrefix(idx, idx - direction, morton_codes,
                                        num_triangles, triangle_ids);

    // Do binary search to compute the upper bound for the length of the range
    int lmax = 2;
    while (LongestCommonPrefix(idx, idx + lmax * direction, morton_codes,
                               num_triangles, triangle_ids) > delta_min) {
      lmax *= 2;
    }

    // Use binary search to find the other end.
    int l = 0;
    int divider = 2;
    for (int t = lmax / divider; t >= 1; divider *= 2) {
      if (LongestCommonPrefix(idx, idx + (l + t) * direction, morton_codes,
                              num_triangles, triangle_ids) > delta_min) {
        l = l + t;
      }
      t = lmax / divider;
    }
    int j = idx + l * direction;

    // Find the length of the longest common prefix for the current node
    int node_delta =
        LongestCommonPrefix(idx, j, morton_codes, num_triangles, triangle_ids);
    int s = 0;
    divider = 2;
    // Search for the split position using binary search.
    for (int t = (l + (divider - 1)) / divider; t >= 1; divider *= 2) {
      if (LongestCommonPrefix(idx, idx + (s + t) * direction, morton_codes,
                              num_triangles, triangle_ids) > node_delta) {
        s = s + t;
      }
      t = (l + (divider - 1)) / divider;
    }
    // gamma in the Karras paper
    int split = idx + s * direction + min(direction, 0);

    // Assign the parent and the left, right children for the current node
    BVHNodePtr<T> curr_node = internal_nodes + idx;
    if (min(idx, j) == split) {
      curr_node->left = leaf_nodes + split;
      (leaf_nodes + split)->parent = curr_node;
    } else {
      curr_node->left = internal_nodes + split;
      (internal_nodes + split)->parent = curr_node;
    }
    if (max(idx, j) == split + 1) {
      curr_node->right = leaf_nodes + split + 1;
      (leaf_nodes + split + 1)->parent = curr_node;
    } else {
      curr_node->right = internal_nodes + split + 1;
      (internal_nodes + split + 1)->parent = curr_node;
    }
  }
  return;
}

template <typename T>
__global__ void CreateHierarchy(BVHNodePtr<T> internal_nodes,
                                BVHNodePtr<T> leaf_nodes, int num_triangles,
                                Triangle<T> *triangles, int *triangle_ids,
                                int *atomic_counters) {
  // int idx = blockDim.x * blockIdx.x + threadIdx.x;
  // if (idx >= num_triangles)
  // return;
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < num_triangles;
       idx += blockDim.x * gridDim.x) {

    BVHNodePtr<T> leaf = leaf_nodes + idx;
    // Assign the index to the primitive
    leaf->idx = triangle_ids[idx];

    Triangle<T> tri = triangles[triangle_ids[idx]];
    // Assign the bounding box of the triangle to the leaves
    leaf->bbox = tri.ComputeBBox();
    leaf->tri_ptr = &triangles[triangle_ids[idx]];
    // leaf->tri_ptr = &triangles[idx];

    BVHNodePtr<T> curr_node = leaf->parent;
    int current_idx = curr_node - internal_nodes;

    // Increment the atomic counter
    int curr_counter = atomicAdd(atomic_counters + current_idx, 1);
    while (true) {
      // atomicAdd returns the old value at the specified address. Thus the
      // first thread to reach this point will immediately return
      if (curr_counter == 0)
        break;

      // Calculate the bounding box of the current node as the union of the
      // bounding boxes of its children.
      AABB<T> left_bb = curr_node->left->bbox;
      AABB<T> right_bb = curr_node->right->bbox;
      curr_node->bbox = left_bb + right_bb;
      // If we have reached the root break
      if (curr_node == internal_nodes)
        break;

      // Proceed to the parent of the node
      curr_node = curr_node->parent;
      // Calculate its position in the flat array
      current_idx = curr_node - internal_nodes;
      // Update the visitation counter
      curr_counter = atomicAdd(atomic_counters + current_idx, 1);
    }
  }

  return;
}

template <typename T>
__global__ void copy_to_tensor(T *dest, T *source, int *ids, int num_elements) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < num_elements;
       idx += blockDim.x * gridDim.x) {
    // dest[idx] = source[ids[idx]];
    dest[ids[idx]] = source[idx];
  }
  return;
}

template <typename T, int blockSize = NUM_THREADS>
void buildBVH(BVHNodePtr<T> internal_nodes, BVHNodePtr<T> leaf_nodes,
              Triangle<T> *__restrict__ triangles,
              thrust::device_vector<int> *triangle_ids, int num_triangles,
              int batch_size) {

#if PRINT_TIMINGS == 1
  // Create the CUDA events used to estimate the execution time of each
  // kernel.
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
#endif

  thrust::device_vector<AABB<T>> bounding_boxes(num_triangles);

  int gridSize = (num_triangles + blockSize - 1) / blockSize;
#if PRINT_TIMINGS == 1
  hipEventRecord(start);
#endif
  // Compute the bounding box for all the triangles
#if DEBUG_PRINT == 1
  std::cout << "Start computing triangle bounding boxes" << std::endl;
#endif
  ComputeTriBoundingBoxes<T><<<gridSize, blockSize>>>(
      triangles, num_triangles, bounding_boxes.data().get());
#if PRINT_TIMINGS == 1
  hipEventRecord(stop);
#endif

  cudaCheckError();

#if DEBUG_PRINT == 1
  std::cout << "Finished computing triangle bounding_boxes" << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Compute Triangle Bounding boxes = " << milliseconds << " (ms)"
            << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventRecord(start);
#endif
  // Compute the union of all the bounding boxes
  AABB<T> host_scene_bb = thrust::reduce(
      bounding_boxes.begin(), bounding_boxes.end(), AABB<T>(), MergeAABB<T>());
#if PRINT_TIMINGS == 1
  hipEventRecord(stop);
#endif

  cudaCheckError();

#if DEBUG_PRINT == 1
  std::cout << "Finished Calculating scene Bounding Box" << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Scene bounding box reduction = " << milliseconds << " (ms)"
            << std::endl;
#endif

  // TODO: Custom reduction ?
  // Copy the bounding box back to the GPU
  AABB<T> *scene_bb_ptr;
  hipMalloc(&scene_bb_ptr, sizeof(AABB<T>));
  hipMemcpy(scene_bb_ptr, &host_scene_bb, sizeof(AABB<T>),
             hipMemcpyHostToDevice);

  thrust::device_vector<MortonCode> morton_codes(num_triangles);
#if DEBUG_PRINT == 1
  std::cout << "Start Morton Code calculation ..." << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventRecord(start);
#endif
  // Compute the morton codes for the centroids of all the primitives
  ComputeMortonCodes<T><<<gridSize, blockSize>>>(
      triangles, num_triangles, scene_bb_ptr, morton_codes.data().get());
#if PRINT_TIMINGS == 1
  hipEventRecord(stop);
#endif

  cudaCheckError();

#if DEBUG_PRINT == 1
  std::cout << "Finished calculating Morton Codes ..." << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Morton code calculation = " << milliseconds << " (ms)"
            << std::endl;
#endif

#if DEBUG_PRINT == 1
  std::cout << "Creating triangle ID sequence" << std::endl;
#endif
  // Construct an array of triangle ids.
  thrust::sequence(triangle_ids->begin(), triangle_ids->end());
#if DEBUG_PRINT == 1
  std::cout << "Finished creating triangle ID sequence ..." << std::endl;
#endif

  // Sort the triangles according to the morton code
#if DEBUG_PRINT == 1
  std::cout << "Starting Morton Code sorting!" << std::endl;
#endif

  try {
#if PRINT_TIMINGS == 1
    hipEventRecord(start);
#endif
    thrust::sort_by_key(morton_codes.begin(), morton_codes.end(),
                        triangle_ids->begin());
#if PRINT_TIMINGS == 1
    hipEventRecord(stop);
#endif
#if DEBUG_PRINT == 1
    std::cout << "Finished morton code sorting!" << std::endl;
#endif
#if PRINT_TIMINGS == 1
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Morton code sorting = " << milliseconds << " (ms)"
              << std::endl;
#endif
  } catch (thrust::system_error e) {
    std::cout << "Error inside Morton code sort: " << e.what() << std::endl;
  }

#if DEBUG_PRINT == 1
  std::cout << "Start building radix tree" << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventRecord(start);
#endif
  // Construct the radix tree using the sorted morton code sequence
  BuildRadixTree<T><<<gridSize, blockSize>>>(
      morton_codes.data().get(), num_triangles, triangle_ids->data().get(),
      internal_nodes, leaf_nodes);
#if PRINT_TIMINGS == 1
  hipEventRecord(stop);
#endif

  cudaCheckError();

#if DEBUG_PRINT == 1
  std::cout << "Finished radix tree" << std::endl;
#endif
#if PRINT_TIMINGS == 1
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Building radix tree = " << milliseconds << " (ms)" << std::endl;
#endif
  // Create an array that contains the atomic counters for each node in the
  // tree
  thrust::device_vector<int> counters(num_triangles);

#if DEBUG_PRINT == 1
  std::cout << "Start Linear BVH generation" << std::endl;
#endif
  // Build the Bounding Volume Hierarchy in parallel from the leaves to the
  // root
  CreateHierarchy<T><<<gridSize, blockSize>>>(
      internal_nodes, leaf_nodes, num_triangles, triangles,
      triangle_ids->data().get(), counters.data().get());

  cudaCheckError();

#if PRINT_TIMINGS == 1
  hipEventRecord(stop);
#endif
#if DEBUG_PRINT == 1
  std::cout << "Finished with LBVH generation ..." << std::endl;
#endif

#if PRINT_TIMINGS == 1
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Hierarchy generation = " << milliseconds << " (ms)"
            << std::endl;
#endif

  hipFree(scene_bb_ptr);
  return;
}


void bvh_distance_queries_kernel(
    const torch::Tensor &triangles, const torch::Tensor &points,
    torch::Tensor *distances, torch::Tensor *closest_points,
    torch::Tensor *closest_faces, torch::Tensor *closest_bcs,
    int queue_size = 128, bool sort_points_by_morton = true) {

  const auto batch_size = triangles.size(0);
  const auto num_triangles = triangles.size(1);
  const auto num_points = points.size(1);

  thrust::device_vector<int> triangle_ids(num_triangles);

  int blockSize = NUM_THREADS;

  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  int gridSize = std::min(
      32 * numSMs, static_cast<int>((num_points + blockSize - 1) / blockSize));

  // Construct the bvh tree
  AT_DISPATCH_FLOATING_TYPES(
      triangles.type(), "bvh_tree_building", ([&] {
      // using scalar_t = float;

#if PRINT_TIMINGS == 1
        // Create the CUDA events used to estimate the execution time of each
        // kernel.
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
#endif

        scalar_t *distances_ptr;
        hipMalloc((void **)&distances_ptr, num_points * sizeof(scalar_t));
        cudaCheckError();

        vec3<scalar_t> *morton_sorted_points_ptr;
        hipMalloc((void **)&morton_sorted_points_ptr,
                   num_points * sizeof(vec3<scalar_t>));
        cudaCheckError();

        vec3<scalar_t> *closest_points_ptr;
        hipMalloc((void **)&closest_points_ptr,
                   num_points * sizeof(vec3<scalar_t>));
        cudaCheckError();

        long *closest_faces_ptr;
        hipMalloc((void **)&closest_faces_ptr, num_points * sizeof(long));
        cudaCheckError();

        vec3<scalar_t> *closest_bcs_ptr;
        hipMalloc((void **)&closest_bcs_ptr, num_points * sizeof(vec3<scalar_t>));
        cudaCheckError();

        // The thrust vectors that contain the BVH nodes
        thrust::device_vector<BVHNode<scalar_t>> leaf_nodes(num_triangles);
        thrust::device_vector<BVHNode<scalar_t>> internal_nodes(num_triangles -
                                                                1);

        auto triangle_scalar_t_ptr = triangles.data<scalar_t>();

        for (int bidx = 0; bidx < batch_size; ++bidx) {

          Triangle<scalar_t> *triangles_ptr =
              (TrianglePtr<scalar_t>)triangle_scalar_t_ptr +
              num_triangles * bidx;

#if DEBUG_PRINT == 1
          std::cout << "Start building BVH" << std::endl;
#endif
          buildBVH<scalar_t, NUM_THREADS>(
              internal_nodes.data().get(), leaf_nodes.data().get(),
              triangles_ptr, &triangle_ids, num_triangles, batch_size);
#if DEBUG_PRINT == 1
          std::cout << "Successfully built BVH" << std::endl;
#endif
          cudaCheckError();

#if DEBUG_PRINT == 1
          std::cout << "Start BVH traversal" << std::endl;
#endif
          vec3<scalar_t> *points_ptr =
              (vec3<scalar_t> *)points.data<scalar_t>() + num_points * bidx;
          thrust::device_vector<int> point_ids(num_points);
          thrust::sequence(point_ids.begin(), point_ids.end());

          if (sort_points_by_morton) {
            thrust::device_vector<MortonCode> morton_codes(num_points);

#if PRINT_TIMINGS == 1
            hipEventRecord(start);
#endif
            ComputePointMortonCodes<scalar_t><<<gridSize, NUM_THREADS>>>(
                // morton_sorted_points.data().get(), points_ptr, num_points,
                morton_sorted_points_ptr, points_ptr, num_points,
                morton_codes.data().get());
            cudaCheckError();
            hipDeviceSynchronize();
#if PRINT_TIMINGS == 1
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Compute morton codes for input points = "
                      << milliseconds << " (ms)" << std::endl;
#endif

            thrust::device_ptr<vec3<scalar_t>> dev_ptr =
                thrust::device_pointer_cast(morton_sorted_points_ptr);

            thrust::sort_by_key(morton_codes.begin(), morton_codes.end(),
                                thrust::make_zip_iterator(thrust::make_tuple(
                                    point_ids.begin(), dev_ptr)));
            cudaCheckError();

            points_ptr = morton_sorted_points_ptr;
          }

#ifdef BVH_PROFILING
          hipProfilerStart();
#endif
          if (queue_size == 32) {
            findNearestNeighbor<scalar_t, 32><<<gridSize, NUM_THREADS>>>(
                points_ptr, distances_ptr, closest_points_ptr,
                closest_faces_ptr, closest_bcs_ptr,
                internal_nodes.data().get(), num_points);
          } else if (queue_size == 64) {
            findNearestNeighbor<scalar_t, 64><<<gridSize, NUM_THREADS>>>(
                points_ptr, distances_ptr, closest_points_ptr,
                closest_faces_ptr, closest_bcs_ptr,
                internal_nodes.data().get(), num_points);
          } else if (queue_size == 128) {
            findNearestNeighbor<scalar_t, 128><<<gridSize, NUM_THREADS>>>(
                points_ptr, distances_ptr, closest_points_ptr,
                closest_faces_ptr, closest_bcs_ptr,
                internal_nodes.data().get(), num_points);
          } else if (queue_size == 256) {
            findNearestNeighbor<scalar_t, 256><<<gridSize, NUM_THREADS>>>(
                points_ptr, distances_ptr, closest_points_ptr,
                closest_faces_ptr, closest_bcs_ptr,
                internal_nodes.data().get(), num_points);
          } else if (queue_size == 512) {
            findNearestNeighbor<scalar_t, 512><<<gridSize, NUM_THREADS>>>(
                points_ptr, distances_ptr, closest_points_ptr,
                closest_faces_ptr, closest_bcs_ptr,
                internal_nodes.data().get(), num_points);
          } else if (queue_size == 1024) {
            findNearestNeighbor<scalar_t, 1024><<<gridSize, NUM_THREADS>>>(
                points_ptr, distances_ptr, closest_points_ptr,
                closest_faces_ptr, closest_bcs_ptr,
                internal_nodes.data().get(), num_points);
          }
          cudaCheckError();
#ifdef BVH_PROFILING
          hipProfilerStop();
#endif

          scalar_t *distances_dest_ptr =
              (scalar_t *)distances->data<scalar_t>() + num_points * bidx;
          vec3<scalar_t> *closest_points_dest_ptr =
              (vec3<scalar_t> *)closest_points->data<scalar_t>() +
              num_points * bidx;
          vec3<scalar_t> *closest_bcs_dest_ptr =
              (vec3<scalar_t> *)closest_bcs->data<scalar_t>() + num_points * bidx;
          long *closest_faces_dest_ptr =
              closest_faces->data<long>() + num_points * bidx;
          if (sort_points_by_morton) {
            copy_to_tensor<scalar_t>
                <<<gridSize, NUM_THREADS>>>(distances_dest_ptr, distances_ptr,
                                            point_ids.data().get(), num_points);
            copy_to_tensor<vec3<scalar_t>><<<gridSize, NUM_THREADS>>>(
                closest_points_dest_ptr, closest_points_ptr,
                point_ids.data().get(), num_points);
            copy_to_tensor<vec3<scalar_t> ><<<gridSize, NUM_THREADS>>>(
                closest_bcs_dest_ptr, closest_bcs_ptr,
                point_ids.data().get(), num_points);
            copy_to_tensor<long><<<gridSize, NUM_THREADS>>>(
                closest_faces_dest_ptr, closest_faces_ptr,
                point_ids.data().get(), num_points);
          } else {
            hipMemcpy(distances_dest_ptr, distances_ptr,
                       num_points * sizeof(scalar_t), hipMemcpyDeviceToDevice);
            hipMemcpy(closest_points_dest_ptr, closest_points_ptr,
                       num_points * sizeof(vec3<scalar_t>),
                       hipMemcpyDeviceToDevice);
            hipMemcpy(closest_bcs_dest_ptr, closest_bcs_ptr,
                       num_points * sizeof(vec3<scalar_t>), hipMemcpyDeviceToDevice);
            hipMemcpy(closest_faces_dest_ptr, closest_faces_ptr,
                       num_points * sizeof(long), hipMemcpyDeviceToDevice);
          }

#if DEBUG_PRINT == 1
          std::cout << "Successfully finished BVH traversal" << std::endl;
#endif
        }
        hipFree(distances_ptr);
        hipFree(closest_points_ptr);
        hipFree(closest_faces_ptr);
        hipFree(closest_bcs_ptr);
        hipFree(morton_sorted_points_ptr);
      }));
}
